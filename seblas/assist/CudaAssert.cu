//
// Created by DanielSun on 4/16/2022.
//

#include "CudaAssert.cuh"

using namespace std;
namespace seblas{
    void assertCuda(const char* file, int line){
        hipError_t error = hipGetLastError();
        if(error != hipSuccess){
            logFatal(seio::LOG_SEG_SEBLAS, string("Device error encountered:") + hipGetErrorString(error));
            logFatal(seio::LOG_SEG_SEBLAS, "line: " + to_string(line) + "  file: " + string(file));
            throw runtime_error("line: " + to_string(line) + "  file: " + string(file));
        }
    }
}