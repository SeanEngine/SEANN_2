#include "hip/hip_runtime.h"
//
// Created by DanielSun on 3/14/2022.
//

#include <utility>
#include "LogUtils.cuh"

namespace seio {
    void printLogHead(LogLevel level, LogSegments segment){

        dye::colorful<basic_string<char>> segName;
        switch(segment){
            case LogSegments::LOG_SEG_SEANN:
                segName = dye::bright_white("seann");
                break;
            case LogSegments::LOG_SEG_SEIO:
                segName = dye::light_green("seio");
                break;
            case LogSegments::LOG_SEG_SEBLAS:
                segName = dye::light_yellow("seblas");
                break;
        }

        dye::colorful<basic_string<char>> levelPrefix;
        switch(level){
            case LogLevel::LOG_LEVEL_DEBUG:
                levelPrefix = dye::purple("DEBUG");
                break;
            case LogLevel::LOG_LEVEL_INFO:
                levelPrefix = dye::light_blue("INFO");
                break;
            case LogLevel::LOG_LEVEL_WARN:
                levelPrefix = dye::light_yellow("WARN");
                break;
            case LogLevel::LOG_LEVEL_ERROR:
                levelPrefix = dye::red("ERROR");
                break;
            case LogLevel::LOG_LEVEL_FATAL:
                levelPrefix = dye::red("FATAL");
                break;
        }

        time_t secs = time(nullptr);
        struct tm *local = localtime(&secs);

        //print the current time
        cout<<dye::light_purple("[")<<dye::light_red(local->tm_hour)<<dye::light_purple(":")
        <<dye::light_red(local->tm_min)<<dye::light_purple(":")<<dye::light_red(local->tm_sec);

        //print the log segment
        cout<<dye::light_purple("|")<<segName<<dye::light_purple("]");

        //print the log level
        if(level == LogLevel::LOG_LEVEL_ERROR || level == LogLevel::LOG_LEVEL_FATAL)
            cout<<dye::red(": ")<<levelPrefix<<dye::red(" >>> ");
        else
            cout<<dye::light_purple(": ")<<levelPrefix<<dye::purple(" >>> ");
    }

    void printColored(const string& msg, LogColor color){
        switch(color){
            case LOG_COLOR_RED:
                cout<<dye::red(msg);
                break;
            case LOG_COLOR_GREEN:
                cout<<dye::green(msg);
                break;
            case LOG_COLOR_YELLOW:
                cout<<dye::yellow(msg);
                break;
            case LOG_COLOR_BLUE:
                cout<<dye::blue(msg);
                break;
            case LOG_COLOR_PURPLE:
                cout<<dye::purple(msg);
                break;
            case LOG_COLOR_AQUA:
                cout<<dye::aqua(msg);
                break;
            case LOG_COLOR_WHITE:
                cout<<dye::white(msg);
                break;
            case LOG_COLOR_LIGHT_RED:
                cout<<dye::light_red(msg);
                break;
            case LOG_COLOR_LIGHT_GREEN:
                cout<<dye::light_green(msg);
                break;
            case LOG_COLOR_LIGHT_YELLOW:
                cout<<dye::light_yellow(msg);
                break;
            case LOG_COLOR_LIGHT_BLUE:
                cout<<dye::light_blue(msg);
                break;
            case LOG_COLOR_LIGHT_PURPLE:
                cout<<dye::light_purple(msg);
                break;
            case LOG_COLOR_LIGHT_AQUA:
                cout<<dye::light_aqua(msg);
                break;
            case LOG_COLOR_BRIGHT_WHITE:
                cout<<dye::bright_white(msg);
                break;
            default:
                cout<<msg;
                break;
        }
    }

    void logInfo(LogSegments segment, string msg){
        printLogHead(LogLevel::LOG_LEVEL_INFO, segment);
        cout<<dye::blue(std::move(msg))<<endl;
    }

    void logInfo(LogSegments seg, const string& msg, LogColor color){
        printLogHead(LogLevel::LOG_LEVEL_INFO, seg);
        printColored(msg, color);
        cout<<endl;
    }

    void logDebug(LogSegments seg, string msg){
        printLogHead(LogLevel::LOG_LEVEL_DEBUG, seg);
        cout<<dye::grey(std::move(msg))<<endl;
    }

    void logDebug(LogSegments seg, const string& msg, LogColor color){
        printLogHead(LogLevel::LOG_LEVEL_DEBUG, seg);
        printColored(msg, color);
        cout<<endl;
    }

    void logWarn(LogSegments seg, string msg){
        printLogHead(LogLevel::LOG_LEVEL_WARN, seg);
        cout<<dye::yellow(std::move(msg))<<endl;
    }

    void logError(LogSegments seg, string msg){
        printLogHead(LogLevel::LOG_LEVEL_ERROR, seg);
        cout<<dye::red(std::move(msg))<<endl;
    }

    void logFatal(LogSegments seg, string msg){
        printLogHead(LogLevel::LOG_LEVEL_FATAL, seg);
        cout<<dye::red(std::move(msg))<<endl;
    }

    #define RENDER_LENGTH 30
    void logProc(unsigned int proc, unsigned int finish) {
        char procBar[RENDER_LENGTH+1] = {0};
        unsigned int procLen = (proc * 30) / finish;
        for(unsigned int i = 0; i < RENDER_LENGTH; i++)
            if (i < procLen)
                procBar[i] = '=';
            else if (i == procLen)
                procBar[i] = '>';
            else
                procBar[i] = ' ';
        procBar[RENDER_LENGTH] = '\0';
        string procBarStr = '[' + string(procBar) + "] ";
        cout<<dye::red(procBarStr)<<dye::light_yellow(to_string(proc))<<
             dye::red("/")<<dye::yellow(to_string(finish));
    }

    void logTrainingProcess(unsigned int batchId, unsigned int epochId, unsigned int batches
        , unsigned int epochs, float loss, float acc, float epochLoss, float epochAcc) {
        cout<<endl;
        printLogHead(LOG_LEVEL_INFO, LOG_SEG_SEANN);
        cout<<dye::purple("Current Batch: ");
        logProc(batchId, batches);
        cout<<"\n     "<<dye::blue("Loss: " + to_string(loss))<<" "<<dye::blue("Acc: " + to_string(acc))<<endl;

        printLogHead(LOG_LEVEL_INFO, LOG_SEG_SEANN);
        cout<<dye::purple("Current Epoch: ");
        logProc(epochId, epochs);
        cout<<"\n     "<<dye::blue("Epoch_Loss: " + to_string(epochLoss))
            <<" "<<dye::blue("Epoch_Acc: " + to_string(epochAcc))<<endl;
    }
}